#include "PDBCUDAUtility.h"
#define NUM_THREADS 128

void printCudaVersion() {
    std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;
    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

void copyFromHostToDevice(void **targetDevice, void *sourceHost, size_t bytesNum) {
  checkCudaErrors(hipMalloc((void **) targetDevice, bytesNum));
  checkCudaErrors(hipMemcpy(*targetDevice, sourceHost, bytesNum, hipMemcpyHostToDevice));
}

void copyFromDeviceToHost(void *targetHost, void * sourceDevice, size_t bytesNum) {
  checkCudaErrors(hipMemcpy(targetHost, sourceDevice, bytesNum, hipMemcpyDeviceToHost));
}

void freeGPUMemory(void ** memdata){
    checkCudaErrors(hipFree(*memdata));
}

void initGPUMemoryToZero(void **memdata, size_t bytesNum) {
    checkCudaErrors(hipMalloc((void **) memdata, bytesNum));
    checkCudaErrors(hipMemset(*memdata, 0, bytesNum));
}

void launchKernel(float *in1data,
                  unsigned int in1NumRow,
                  unsigned int in1NumCol,
                  float *in2data,
                  unsigned int in2NumRow,
                  unsigned int in2NumCol,
                  float *outdataGPU) {
    hipblasHandle_t handle;
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    checkCudaErrors(hipblasCreate(&handle));
    checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, in2NumCol, in1NumRow, in1NumCol, &alpha, in2data, in2NumCol, in1data, in1NumCol, &beta, outdataGPU, in2NumCol));
    //matrixMulGPU <<< number_of_blocks, threads_per_block >>> (in1data, in1NumRow, in1NumCol, in2data, in2NumRow, in2NumCol, outdataGPU);
}




