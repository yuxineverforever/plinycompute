#include <assert.h>
#include "PDBCUDAUtility.h"

void printCudaVersion() {
    std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;
    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

void copyFromHostToDevice(void **targetDevice, void *sourceHost, size_t bytesNum) {
    checkCudaErrors(hipMalloc((void **) targetDevice, bytesNum));
    checkCudaErrors(hipMemcpy(*targetDevice, sourceHost, bytesNum, hipMemcpyHostToDevice));
}

void copyFromHostToDeviceAsync(void **targetDevice, void *sourceHost, size_t bytesNum, hipStream_t cs) {
    checkCudaErrors(hipMalloc((void **) targetDevice, bytesNum));
    checkCudaErrors(hipMemcpyAsync((*targetDevice), sourceHost, bytesNum, hipMemcpyHostToDevice, cs));
}

void copyFromDeviceToHost(void *targetHost, void *sourceDevice, size_t bytesNum) {
    checkCudaErrors(hipMemcpy(targetHost, sourceDevice, bytesNum, hipMemcpyDeviceToHost));
}

void copyFromDeviceToHostAsync(void *targetHost, void *sourceDevice, size_t bytesNum, hipStream_t cs) {
    checkCudaErrors(hipMemcpyAsync(targetHost, sourceDevice, bytesNum, hipMemcpyDeviceToHost, cs));
}

void freeGPUMemory(void **memdata) {
    checkCudaErrors(hipFree(*memdata));
}

void initGPUMemoryToZero(void **memdata, size_t bytesNum) {
    checkCudaErrors(hipMalloc((void **) memdata, bytesNum));
    checkCudaErrors(hipMemset(*memdata, 0, bytesNum));
}

int isDevicePointer(const void *ptr) {
    hipPointerAttribute_t attributes;
    hipPointerGetAttributes(&attributes, ptr);
    hipError_t err = hipGetLastError();
    if (attributes.devicePointer != nullptr && err == 0) {
        return 1;
    } else {
        return 0;
    }
}



