#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include "PDBCUDAMatrixMultiple.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#define NUM_THREADS 128

__global__ void matrixMulGPU(float *in1data,
                             unsigned int in1NumRow,
                             unsigned int in1NumCol,
                             float *in2data,
                             unsigned int in2NumRow,
                             unsigned int in2NumCol,
                             float *outdata) {
  if (in1NumCol != in2NumRow) {
    return;
  }
  unsigned int I = in1NumRow;
  unsigned int J = in2NumCol;
  unsigned int K = in1NumCol;

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < I && col < J) {
    for (int k = 0; k < K; ++k) {
      outdata[row * J + col] += in1data[row * K + k] * in2data[k * J + col];
    }
  }
}

void copyFromHostToDevice(float **targetDevice, float *sourceHost, unsigned int numRows, unsigned int numCols) {
  const unsigned int numElems = numRows * numCols;
  checkCudaErrors(hipMalloc((void **) targetDevice, numElems * sizeof(float)));
  checkCudaErrors(hipMemcpy(*targetDevice, sourceHost, numElems * sizeof(float), hipMemcpyHostToDevice));
}

void copyFromDeviceToHost(float *targetHost, float *sourceDevice, unsigned int numRows, unsigned int numCols) {
  const unsigned int numElems = numRows * numCols;
  checkCudaErrors(hipMemcpy(targetHost, sourceDevice, numElems * sizeof(float), hipMemcpyDeviceToHost));
}

void launchKernel(float *in1data,
                  unsigned int in1NumRow,
                  unsigned int in1NumCol,
                  float *in2data,
                  unsigned int in2NumRow,
                  unsigned int in2NumCol,
                  float *outdataGPU) {
    hipblasHandle_t handle;
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    dim3 threads_per_block(NUM_THREADS, NUM_THREADS, 1);
    dim3 number_of_blocks((in1NumRow / threads_per_block.x) + 1, (in2NumCol / threads_per_block.y) + 1, 1);
    checkCudaErrors(hipblasCreate(&handle));
    checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, in2NumCol, in1NumRow, in1NumCol, &alpha, in2data, in2NumCol, in1data, in1NumCol, &beta, outdataGPU, in2NumCol));
    //matrixMulGPU <<< number_of_blocks, threads_per_block >>> (in1data, in1NumRow, in1NumCol, in2data, in2NumRow, in2NumCol, outdataGPU);
}

void initGPUMemoryToZero(float **memdata, unsigned int numRows, unsigned int numCols) {
  const unsigned int numElems = numRows * numCols;
  checkCudaErrors(hipMalloc((void **) memdata, numElems * sizeof(float)));
  checkCudaErrors(hipMemset(*memdata, 0, numElems * sizeof(float)));
}

void printCudaVersion() {
  std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;
  int runtime_ver;
  hipRuntimeGetVersion(&runtime_ver);
  std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;
  int driver_ver;
  hipDriverGetVersion(&driver_ver);
  std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

void freeGPUMemory(float ** memdata){
  checkCudaErrors(hipFree(*memdata));
}
